#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "image_enhancement.cu.h"

#define LUT_RES 256
#define EPSILON 1.0f / 256.0f


__device__ float to_gray(float rgb[3]){
    return rgb[0] * 0.2125f + rgb[1] * 0.7154f + rgb[2] * 0.0721f;
}

extern "C"
__global__ void color_to_gray(uint8_t* color, float* gray, uint32_t width, uint32_t height){
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (width <= x || height <= y){
        return;
    }

    float rgb[3];
    rgb[0] = ((float) color[y * width * 3 + x * 3 + 0]) / 255.0f;
    rgb[1] = ((float) color[y * width * 3 + x * 3 + 1]) / 255.0f;
    rgb[2] = ((float) color[y * width * 3 + x * 3 + 2]) / 255.0f;

    gray[y * width + x] = to_gray(rgb);
}

extern "C"
__global__ void photometric_mask_ud(float* ph_mask, float* lut, uint32_t width, uint32_t height){
    uint32_t j = blockIdx.x * blockDim.x + threadIdx.x;

    if (width <= j){
        return;
    }

    for (uint32_t i = 0; i < height - 2; ++i){
        float diff = abs(ph_mask[i * width + j] - ph_mask[(i + 2) * width + j]);
        float sigmoid = lut[(uint8_t)(diff * (LUT_RES - 1))];
        ph_mask[(i + 1) * width + j] = ph_mask[(i + 1) * width + j] * sigmoid + ph_mask[i * width + j] * (1 - sigmoid);
    }
}

extern "C"
__global__ void photometric_mask_du(float* ph_mask, float* lut, uint32_t width, uint32_t height){
    uint32_t j = blockIdx.x * blockDim.x + threadIdx.x;

    if (width <= j){
        return;
    }

    for (uint32_t i = height - 2; i > 1; --i){
        float diff = abs(ph_mask[(i - 1) * width + j] - ph_mask[(i + 1) * width + j]);
        float sigmoid = lut[(uint8_t)(diff * (LUT_RES - 1))];
        ph_mask[i * width + j] = ph_mask[i * width + j] * sigmoid + ph_mask[(i + 1) * width + j] * (1 - sigmoid);
    }
}

extern "C"
__global__ void photometric_mask_lr(float* ph_mask, float* lut, uint32_t width, uint32_t height){
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (height <= i){
        return;
    }

    for (uint32_t j = 0; j < width - 2; ++j){
        float diff = abs(ph_mask[i * width + j] - ph_mask[i * width + j + 2]);
        float sigmoid = lut[(uint8_t)(diff * (LUT_RES - 1))];
        ph_mask[i * width + j + 1] = ph_mask[i * width + j + 1] * sigmoid + ph_mask[i * width + j] * (1 - sigmoid);
    }
}

extern "C"
__global__ void photometric_mask_rl(float* ph_mask, float* lut, uint32_t width, uint32_t height){
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (height <= i){
        return;
    }

    for (uint32_t j = width - 2; j > 1; --j){
        float diff = abs(ph_mask[i * width + j - 1] - ph_mask[i * width + j + 1]);
        float sigmoid = lut[(uint8_t)(diff * (LUT_RES - 1))];
        ph_mask[i * width + j] = ph_mask[i * width + j] * sigmoid + ph_mask[i * width + j + 1] * (1 - sigmoid);
    }
}

__device__ float local_contrast_enhancement(
    float gray, float mask, float threshold_dark_tones, float local_boost,
    float detail_amp_global
){
    float detail_amp_local = min(mask / threshold_dark_tones, 1.0f);
    detail_amp_local = (1 - detail_amp_local) * local_boost + 1;

    float value = mask + (gray - mask) * detail_amp_global * detail_amp_local;

    return max(0.0f, min(1.0f, value));
}

__device__ float spatial_tonemapping(
    float value, float mask, float mid_tone_mapped, float tonal_width_mapped, float areas_dark_mapped,
    float areas_bright_mapped
){
    float alpha;
    float tone_continuation_factor;

    float mask_inv = 1 - mask;

    float lower = value * (value < mid_tone_mapped);
    float upper = value * (value >= mid_tone_mapped);

    alpha = (mask * mask) / tonal_width_mapped;
    tone_continuation_factor = mid_tone_mapped / (mid_tone_mapped + EPSILON - mask);
    alpha = alpha * tone_continuation_factor + areas_dark_mapped;
    lower = (lower * (alpha + 1)) / (alpha + lower);

    alpha = (mask_inv * mask_inv) / tonal_width_mapped;
    tone_continuation_factor = mid_tone_mapped / ((1 - mid_tone_mapped) - mask_inv);
    alpha = alpha * tone_continuation_factor + areas_bright_mapped;
    upper = (upper * alpha) / (alpha + 1 - upper);

    return lower + upper;
}

__device__ float srgb_to_linear(float value){
    float lower = value * (value <= 0.04045f) / 12.92f;
    float upper = powf((value + 0.055f) * (value > 0.04045f) / 1.055f, 2.4f);

    return lower + upper;
}

__device__ float linear_to_srgb(float value){
    float lower = value * (value <= 0.0031308f) * 12.92f;
    float upper = powf(value * (value > 0.0031308f), 1.0f / 2.4f) * 1.055f - 0.055f;

    return max(0.0f, min(1.0f, lower + upper));
}

__device__ void graytone_to_color(float rgb[3], float gray){
    rgb[0] = srgb_to_linear(rgb[0]);
    rgb[1] = srgb_to_linear(rgb[1]);
    rgb[2] = srgb_to_linear(rgb[2]);

    float graytone_linear = srgb_to_linear(gray);

    float gray_linear = to_gray(rgb);
    if (gray_linear <= 0.0f){
        gray_linear = EPSILON;
    }

    float tone_ratio = graytone_linear / gray_linear;

    rgb[0] = max(0.0f, min(1.0f, rgb[0] * tone_ratio));
    rgb[1] = max(0.0f, min(1.0f, rgb[1] * tone_ratio));
    rgb[2] = max(0.0f, min(1.0f, rgb[2] * tone_ratio));

    rgb[0] = linear_to_srgb(rgb[0]);
    rgb[1] = linear_to_srgb(rgb[1]);
    rgb[2] = linear_to_srgb(rgb[2]);
}

__device__ void change_color_saturation(
    float rgb[3], float mask, float threshold_dark_tones, float local_boost, float saturation_degree
){
    float gray = (rgb[0] + rgb[1] + rgb[2]) / 3.0f;

    rgb[0] = rgb[0] - gray;
    rgb[1] = rgb[1] - gray;
    rgb[2] = rgb[2] - gray;

    float detail_amplification_local = ((1 - min(1.0f, mask / threshold_dark_tones)) * local_boost) + 1;

    rgb[0] = max(0.0f, min(1.0f, gray + rgb[0] * saturation_degree * detail_amplification_local));
    rgb[1] = max(0.0f, min(1.0f, gray + rgb[1] * saturation_degree * detail_amplification_local));
    rgb[2] = max(0.0f, min(1.0f, gray + rgb[2] * saturation_degree * detail_amplification_local));
}

extern "C"
__global__ void enhance_image(
    uint8_t* image, float* ph_mask, float threshold_dark_tones, float local_boost, float saturation_degree,
    float mid_tone_mapped, float tonal_width_mapped, float areas_dark_mapped, float areas_bright_mapped, float detail_amp_global, uint32_t width, uint32_t height
){
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (width <= x || height <= y){
        return;
    }

    float rgb[3];
    rgb[0] = ((float) image[y * width * 3 + x * 3 + 0]) / 255.0f;
    rgb[1] = ((float) image[y * width * 3 + x * 3 + 1]) / 255.0f;
    rgb[2] = ((float) image[y * width * 3 + x * 3 + 2]) / 255.0f;

    float mask = ph_mask[y * width + x];

    float gray;
    gray = to_gray(rgb);
    gray = local_contrast_enhancement(gray, mask, threshold_dark_tones, local_boost, detail_amp_global);
    gray = spatial_tonemapping(
        gray, mask, mid_tone_mapped, tonal_width_mapped, areas_dark_mapped,
        areas_bright_mapped
    );

    graytone_to_color(rgb, gray);

    change_color_saturation(rgb, mask, threshold_dark_tones, local_boost, saturation_degree);

    image[y * width * 3 + x * 3 + 0] = (uint8_t) max(0.0f, min(255.0f, rgb[0] * 255.0f));
    image[y * width * 3 + x * 3 + 1] = (uint8_t) max(0.0f, min(255.0f, rgb[1] * 255.0f));
    image[y * width * 3 + x * 3 + 2] = (uint8_t) max(0.0f, min(255.0f, rgb[2] * 255.0f));
}
